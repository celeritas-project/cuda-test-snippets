#include "hip/hip_runtime.h"
#include "corecel/device_runtime_api.h"
#include "corecel/Assert.hh"
#include "corecel/Types.hh"
#include "corecel/sys/Device.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"

#include "celeritas/global/alongstep/AlongStepLauncher.hh"
#include "celeritas/global/alongstep/detail/AlongStepUniformMsc.hh"

using namespace celeritas;

//---------------------------------------------------------------------------//
__global__ void
along_step_uniform_msc_kernel(DeviceCRef<CoreParamsData> const params,
                              DeviceRef<CoreStateData> const state,
                              DeviceCRef<UrbanMscData> const msc_data,
                              UniformFieldParams const field_params)
{
    auto tid = KernelParamCalculator::thread_id();
    if (!(tid < state.size()))
        return;

    auto launch = make_along_step_launcher(params,
                                           state,
                                           msc_data,
                                           field_params,
                                           NoData{},
                                           detail::along_step_uniform_msc);
    launch(tid);
}
