#include "hip/hip_runtime.h"
#include "base/Array.hh"

using ArrayInt = celeritas::Array<int, 4>;

__global__ void assign(const int* src, ArrayInt* dst)
{
    dst[threadIdx.x].fill(src[threadIdx.x]);
}
