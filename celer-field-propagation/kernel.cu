#include "hip/hip_runtime.h"

#include "celeritas/Types.hh"
#include "celeritas/field/DormandPrinceStepper.hh"
#include "celeritas/field/FieldDriverOptions.hh"
#include "celeritas/field/MakeMagFieldPropagator.hh"
#include "celeritas/field/UniformField.hh"
#include "celeritas/field/UniformFieldData.hh"
#include "celeritas/global/CoreTrackData.hh"
#include "celeritas/global/CoreTrackView.hh"
#include "celeritas/track/SimTrackView.hh"
#include "corecel/Macros.hh"
#include "corecel/Types.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "corecel/sys/ThreadId.hh"

using namespace celeritas;
using namespace celeritas::detail;

__global__ void test_kernel(CoreRef<MemSpace::device> const track_data,
                            UniformFieldParams const field_par) {
  auto tid = KernelParamCalculator::thread_id();
  const celeritas::CoreTrackView track(track_data.params, track_data.states,
                                       tid);
  auto sim = track.make_sim_view();
  auto step_limit = sim.step_limit();

  auto particle = track.make_particle_view();
  auto geo = track.make_geo_view();
  auto propagate = make_mag_field_propagator<DormandPrinceStepper>(
      UniformField(field_par.field), field_par.options, particle, &geo);
  Propagation p = propagate(step_limit.step);
  if (p.boundary) {
    // Stopped at a geometry boundary: this is the new step action.
    CELER_ASSERT(p.distance <= step_limit.step);
    CELER_ASSERT(p.distance < step_limit.step);
    step_limit.step = p.distance;
    step_limit.action = track.boundary_action();
  } else if (p.distance < step_limit.step) {
    // Some other internal non-boundary geometry limit has been reached
    // (e.g. too many substeps)
    step_limit.step = p.distance;
    step_limit.action = track.propagation_limit_action();
  }
  sim.force_step_limit(step_limit);
}
