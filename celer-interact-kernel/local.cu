#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file KNDemoKernel.cu
//---------------------------------------------------------------------------//
// NOTE: this is commit 7b35b03 of the "performance-tweaks" branch.
#include "KNDemoKernel.hh"

#include "base/ArrayUtils.hh"
#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/SecondaryAllocatorView.hh"
#include "physics/em/detail/KleinNishinaInteractor.hh"
#include "random/cuda/RngEngine.hh"
#include "physics/grid/PhysicsGridCalculator.hh"
#include "DetectorView.hh"
#include "KernelUtils.hh"

using namespace celeritas;
using celeritas::detail::KleinNishinaInteractor;

namespace demo_interactor
{
//---------------------------------------------------------------------------//
/*!
 * Perform a single interaction per particle track.
 *
 * The interaction:
 * - Clears the energy deposition
 * - Samples the KN interaction
 * - Allocates and emits a secondary
 * - Kills the secondary, depositing its local energy
 * - Applies the interaction (updating track direction and energy)
 */
__global__ void iterate_kernel(ParamsDeviceRef const            params,
                               StateDeviceRef const             states,
                               SecondaryAllocatorPointers const secondaries,
                               DetectorPointers const           detector)
{
    SecondaryAllocatorView allocate_secondaries(secondaries);
    DetectorView           detector_hit(detector);
    PhysicsGridCalculator  calc_xs(params.tables.xs, params.tables.reals);

    for (unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < states.size();
         tid += blockDim.x * gridDim.x)
    {
        // Skip loop if already dead
        if (!states.alive[tid])
        {
            continue;
        }

        // Construct particle accessor from immutable and thread-local data
        ParticleTrackView particle(
            params.particle, states.particle, ThreadId(tid));
        RngEngine rng(states.rng, ThreadId(tid));
        Real3     pos  = states.position[tid];
        Real3     dir  = states.direction[tid];
        real_type time = states.time[tid];

        // Move to collision
        demo_interactor::move_to_collision(
            particle, calc_xs, dir, &pos, &time, rng);

        if (particle.energy() < units::MevEnergy{0.01})
        {
            // Particle is below interaction energy
            Hit h;
            h.pos              = pos;
            h.thread           = ThreadId(tid);
            h.time             = time;
            h.dir              = dir;
            h.energy_deposited = particle.energy();

            // Deposit energy and kill
            detector_hit(h);
            states.alive[tid] = false;
            continue;
        }

        // Construct RNG and interaction interfaces
        KleinNishinaInteractor interact(
            params.kn_interactor, particle, dir, allocate_secondaries);

        // Perform interaction: should emit a single particle (an electron)
        Interaction interaction = interact(rng);
        CELER_ASSERT(interaction);
        CELER_ASSERT(interaction.secondaries.size() == 1);

        // Deposit energy from the secondary (effectively, an infinite energy
        // cutoff)
        {
            Hit h;
            h.pos                 = pos;
            h.thread              = ThreadId(tid);
            h.time                = time;
            const auto& secondary = interaction.secondaries.front();
            h.dir                 = secondary.direction;
            h.energy_deposited    = secondary.energy;
            detector_hit(h);
        }

        // Update post-interaction state (apply interaction)
        states.direction[tid] = interaction.direction;
        states.position[tid]  = pos;
        states.time[tid]      = time;
        particle.energy(interaction.energy);
    }
}
//---------------------------------------------------------------------------//
} // namespace demo_interactor
