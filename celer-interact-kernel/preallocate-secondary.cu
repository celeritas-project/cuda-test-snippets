#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file KNDemoKernel.cu
//---------------------------------------------------------------------------//
// NOTE: this is commit 9f98601 of the "performance-tweaks" branch.
#include "KNDemoKernel.hh"

#include "base/ArrayUtils.hh"
#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/em/detail/KleinNishinaInteractor.hh"
#include "random/cuda/RngEngine.hh"
#include "physics/grid/PhysicsGridCalculator.hh"
#include "DetectorView.hh"
#include "KernelUtils.hh"

using namespace celeritas;
using celeritas::detail::KleinNishinaInteractor;

namespace demo_interactor
{
//---------------------------------------------------------------------------//
/*!
 * Perform a single interaction per particle track.
 *
 * The interaction:
 * - Clears the energy deposition
 * - Samples the KN interaction
 * - Emits a secondary as part of the interaction
 * - Kills the secondary, depositing its local energy
 * - Applies the interaction (updating track direction and energy)
 */
__global__ void iterate_kernel(ParamsDeviceRef const            params,
                               StateDeviceRef const             states,
                               DetectorPointers const           detector)
{
    DetectorView           detector_hit(detector);
    PhysicsGridCalculator  calc_xs(params.tables.xs, params.tables.reals);

    for (unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < states.size();
         tid += blockDim.x * gridDim.x)
    {
        // Skip loop if already dead
        if (!states.alive[tid])
        {
            continue;
        }

        // Construct particle accessor from immutable and thread-local data
        ParticleTrackView particle(
            params.particle, states.particle, ThreadId(tid));
        RngEngine rng(states.rng, ThreadId(tid));

        // Move to collision
        demo_interactor::move_to_collision(particle,
                                           calc_xs,
                                           states.direction[tid],
                                           &states.position[tid],
                                           &states.time[tid],
                                           rng);

        Hit h;
        h.pos    = states.position[tid];
        h.thread = ThreadId(tid);
        h.time   = states.time[tid];

        if (particle.energy() < units::MevEnergy{0.01})
        {
            // Particle is below interaction energy
            h.dir              = states.direction[tid];
            h.energy_deposited = particle.energy();

            // Deposit energy and kill
            detector_hit(h);
            states.alive[tid] = false;
            continue;
        }

        // Construct RNG and interaction interfaces
        KleinNishinaInteractor interact(
            params.kn_interactor, particle, states.direction[tid]);

        // Perform interaction: should emit a single particle (an electron)
        Interaction interaction = interact(rng);
        CELER_ASSERT(interaction);
        CELER_ASSERT(interaction.num_secondaries() == 1);

        // Deposit energy from the secondary (effectively, an infinite energy
        // cutoff)
        {
            const auto& secondary = interaction.secondary;
            CELER_ASSERT(secondary);
            h.dir                 = secondary.direction;
            h.energy_deposited    = secondary.energy;
            detector_hit(h);
        }

        // Update post-interaction state (apply interaction)
        states.direction[tid] = interaction.direction;
        particle.energy(interaction.energy);
    }
}
//---------------------------------------------------------------------------//
} // namespace demo_interactor
