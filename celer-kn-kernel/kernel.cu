#include "hip/hip_runtime.h"
#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "random/RngEngine.hh"
#include "physics/base/ModelData.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/PhysicsTrackView.hh"
#include "base/StackAllocator.hh"
#include "physics/em/detail/KleinNishinaInteractor.hh"

using namespace celeritas;
using namespace celeritas::detail;

__global__ void test_kernel(const KleinNishinaData kn,
                            const ModelInteractRef<MemSpace::device> model) {
  auto tid = KernelParamCalculator::thread_id();

  StackAllocator<Secondary> allocate_secondaries(model.states.secondaries);
  ParticleTrackView particle(model.params.particle, model.states.particle, tid);

  KleinNishinaInteractor interact(
      kn, particle, model.states.direction[tid], allocate_secondaries);

  RngEngine rng(model.states.rng, tid);
  model.states.interactions[tid] = interact(rng);
}
