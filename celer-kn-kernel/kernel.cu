#include "hip/hip_runtime.h"
#include "base/KernelParamCalculator.device.hh"
#include "sim/CoreTrackData.hh"
#include "random/RngEngine.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/PhysicsTrackView.hh"
#include "base/StackAllocator.hh"
#include "physics/em/detail/KleinNishinaInteractor.hh"

using namespace celeritas;
using namespace celeritas::detail;

__global__ void test_kernel(
    const CoreRef<MemSpace::native> core,
    const KleinNishinaData kn,
    Interaction* results) {
  auto tid = KernelParamCalculator::thread_id();

  StackAllocator<Secondary> allocate_secondaries(core.states.secondaries);
  ParticleTrackView particle(core.params.particles, core.states.particles, tid);

  KleinNishinaInteractor interact(
      kn, particle, core.states.geometry.dir[tid], allocate_secondaries);

  RngEngine rng(core.states.rng, tid);
  results[tid.get()] = interact(rng);
}
