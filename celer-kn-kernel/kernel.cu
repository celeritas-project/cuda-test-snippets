#include "hip/hip_runtime.h"
#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "random/RngEngine.hh"
#include "physics/base/ModelInterface.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/PhysicsTrackView.hh"
#include "base/StackAllocator.hh"
#include "physics/em/detail/KleinNishinaInteractor.hh"

using namespace celeritas;
using namespace celeritas::detail;

__global__ void test_kernel(const KleinNishinaPointers kn,
                            const ModelInteractPointers model) {
  auto tid = celeritas::KernelParamCalculator::thread_id();

  StackAllocator<Secondary> allocate_secondaries(model.secondaries);
  ParticleTrackView particle(model.params.particle, model.states.particle, tid);

  KleinNishinaInteractor interact(kn, particle,
                              model.states.direction[tid.get()],
                              allocate_secondaries);

  RngEngine rng(model.states.rng, tid);
  model.result[tid.get()] = interact(rng);
}
