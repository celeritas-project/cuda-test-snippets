#include "hip/hip_runtime.h"
#include "orange/surfaces/SurfaceAction.test.hh"
#include "base/KernelParamCalculator.cuda.hh"


using namespace celeritas_test;

__global__ void sa_test_kernel(SATestInput input)
{
    auto tid = celeritas::KernelParamCalculator::thread_id();
    if (tid.get() >= input.states.size())
        return;

    // Calculate distances in parallel
    CalcSenseDistanceLauncher<> calc_thread{input.params, input.states};
    calc_thread(tid);
}
