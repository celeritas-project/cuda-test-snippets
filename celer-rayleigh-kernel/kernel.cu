#include "hip/hip_runtime.h"
#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "random/RngEngine.hh"
#include "physics/base/ModelInterface.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/PhysicsTrackView.hh"
#include "base/StackAllocator.hh"
#include "physics/em/detail/RayleighInteractor.hh"

using namespace celeritas;
using namespace celeritas::detail;

__global__ void test_kernel(const RayleighDeviceRef rayleigh,
                            const ModelInteractPointers model) {
  auto tid = celeritas::KernelParamCalculator::thread_id();

  ParticleTrackView particle(model.params.particle, model.states.particle, tid);
  ElementId el_id{0};

  RayleighInteractor interact(rayleigh, particle,
                              model.states.direction[tid.get()], el_id);

  RngEngine rng(model.states.rng, tid);
  model.result[tid.get()] = interact(rng);
}
