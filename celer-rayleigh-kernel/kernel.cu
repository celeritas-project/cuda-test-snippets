#include "hip/hip_runtime.h"
#include "base/Assert.hh"
#include "base/KernelParamCalculator.cuda.hh"
#include "random/RngEngine.hh"
#include "physics/base/ModelData.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/PhysicsTrackView.hh"
#include "base/StackAllocator.hh"
#include "physics/em/detail/RayleighInteractor.hh"

using namespace celeritas;
using namespace celeritas::detail;

__global__ void test_kernel(const RayleighDeviceRef rayleigh,
                            const ModelInteractRef<MemSpace::device> model) {
  auto tid = celeritas::KernelParamCalculator::thread_id();

  ParticleTrackView particle(model.params.particle, model.states.particle, tid);
  ElementId el_id{0};

  RayleighInteractor interact(rayleigh, particle,
                              model.states.direction[tid], el_id);

  RngEngine rng(model.states.rng, tid);
  model.states.interactions[tid] = interact(rng);
}
