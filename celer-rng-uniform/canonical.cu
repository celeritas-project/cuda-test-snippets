#include "hip/hip_runtime.h"
#include "random/RngEngine.hh"
#include "base/KernelParamCalculator.device.hh"

using namespace celeritas;

__global__ void sample(RngEngine::StateRef const states, double* result)
{
    auto thread_id = KernelParamCalculator::thread_id();
    RngEngine rng(states, thread_id);

    result[3 * thread_id.get()] = generate_canonical(rng);
    result[3 * thread_id.get() + 1] = 5 * generate_canonical(rng);
    result[3 * thread_id.get() + 2] = 2.5 + 5.25 * generate_canonical(rng);
}
