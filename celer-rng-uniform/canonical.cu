#include "hip/hip_runtime.h"
#include "random/cuda/RngEngine.hh"
#include "base/KernelParamCalculator.cuda.hh"

using namespace celeritas;

__global__ void sample(RngStatePointers const states, double* result)
{
    auto thread_id = KernelParamCalculator::thread_id();
    RngEngine rng(states, thread_id);

    result[3 * thread_id.get()] = generate_canonical(rng);
    result[3 * thread_id.get() + 1] = 5 * generate_canonical(rng);
    result[3 * thread_id.get() + 2] = 2.5 + 5.25 * generate_canonical(rng);
}
