#include "hip/hip_runtime.h"
#include "random/RngEngine.hh"
#include "random/distributions/UniformRealDistribution.hh"
#include "base/KernelParamCalculator.cuda.hh"

using namespace celeritas;

__device__ __forceinline__ double sample(RngEngine& r, double lower, double
                                         upper)
{
    return UniformRealDistribution<double>(lower, upper)(r);
}

__global__ void sample(RngEngine::StateRef const states, double* result)
{
    auto thread_id = KernelParamCalculator::thread_id();
    RngEngine rng(states, thread_id);

    result[3 * thread_id.get()] = sample(rng, 0, 1);
    result[3 * thread_id.get() + 1] = sample(rng, 0, 5);
    result[3 * thread_id.get() + 2] = sample(rng, 2.5, 7.75);
}
