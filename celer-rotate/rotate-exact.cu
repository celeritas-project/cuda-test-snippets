#include "hip/hip_runtime.h"
#include "base/ArrayUtils.hh"

using namespace celeritas;

__global__ void rotate_kernel(
    Real3 const * const __restrict__ dir,
    Real3 const * const __restrict__ rot,
    Real3 * const __restrict__ dst)
{
    dst[threadIdx.x] = rotate(dir[threadIdx.x], rot[threadIdx.x]);
}
