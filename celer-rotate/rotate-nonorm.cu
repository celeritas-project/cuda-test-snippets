#include "hip/hip_runtime.h"
#include "base/ArrayUtils.hh"

using namespace celeritas;

/*!
 * Rotate the direction about the given Z-based scatter direction.
 *
 * The equivalent to Shift's \code
 * void cartesian_vector_transform(
    double      costheta,
    double      phi,
    Vector_View vector)
 * \endcode
 * is the call
 * \code
   vector = rotate(from_spherical(costheta, phi), vector);
 * \endcode
 *
 * This code effectively decomposes the given rotation vector \c rot into two
 * sequential transform matrices, one with an angle \em theta about the \em y
 * axis and one about \em phi rotating around the \em z axis. These two angles
 * are the spherical coordinate transform of the given \c rot cartesian
 * direction vector.
 *
 * There is some extra code in here to deal with loss of precision when the
 * incident direction is along the \em z axis. As \c rot approaches \em z, the
 * azimuthal angle \em phi must be calculated carefully from both the x and y
 * components of the vector, not independently. If \c rot actually equals \em z
 * then the azimuthal angle is completely indeterminate so we arbitrarily
 * choose \c phi = 0.
 *
 * This function is often used for calculating exiting scattering angles. In
 * that case, \c dir is the exiting angle from the scattering calculation, and
 * \c rot is the original direction of the particle. The direction vectors are
 * defined
 * \f[
   \Omega =   \sin\theta\cos\phi\mathbf{i}
            + \sin\theta\sin\phi\mathbf{j}
            + \cos\theta\mathbf{k}
 * \f]
 */
inline CELER_FUNCTION Real3 rotate_mod(const Real3& dir, const Real3& rot)
{
    CELER_EXPECT(is_soft_unit_vector(dir));
    CELER_EXPECT(is_soft_unit_vector(rot));

    // Direction enumeration
    enum
    {
        X = 0,
        Y = 1,
        Z = 2
    };

    // Transform direction vector into theta, phi so we can use it as a
    // rotation matrix
    real_type sintheta = std::sqrt(1 - ipow<2>(rot[Z]));
    real_type cosphi;
    real_type sinphi;

    if (sintheta >= detail::RealVecTraits<real_type>::min_accurate_sintheta())
    {
        // Typical case: far enough from z axis to assume the X and Y
        // components have a hypotenuse of 1 within epsilon tolerance
        const real_type inv_sintheta = 1 / (sintheta);
        cosphi                       = rot[X] * inv_sintheta;
        sinphi                       = rot[Y] * inv_sintheta;
    }
    else if (sintheta > 0)
    {
        // Avoid catastrophic roundoff error by normalizing x/y components
        cosphi = rot[X] / std::sqrt(ipow<2>(rot[X]) + ipow<2>(rot[Y]));
        sinphi = std::sqrt(1 - ipow<2>(cosphi));
    }
    else
    {
        // NaN or 0: choose an arbitrary azimuthal angle for the incident dir
        cosphi = 1;
        sinphi = 0;
    }

    Real3 result
        = {(rot[Z] * dir[X] + sintheta * dir[Z]) * cosphi - sinphi * dir[Y],
           (rot[Z] * dir[X] + sintheta * dir[Z]) * sinphi + cosphi * dir[Y],
           -sintheta * dir[X] + rot[Z] * dir[Z]};

    return result;
}

__global__ void rotate_kernel(
    Real3 const * const __restrict__ dir,
    Real3 const * const __restrict__ rot,
    Real3 * const __restrict__ dst)
{
    dst[threadIdx.x] = rotate_mod(dir[threadIdx.x], rot[threadIdx.x]);
}
