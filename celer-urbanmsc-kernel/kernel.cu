#include "hip/hip_runtime.h"

#include "celeritas/Types.hh"
#include "celeritas/em/data/UrbanMscData.hh"
#include "celeritas/em/msc/UrbanMsc.hh"
#include "celeritas/global/CoreTrackData.hh"
#include "celeritas/global/CoreTrackView.hh"
#include "celeritas/track/SimTrackView.hh"
#include "corecel/Macros.hh"
#include "corecel/Types.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "corecel/sys/ThreadId.hh"

using namespace celeritas;
using namespace celeritas::detail;

__global__ void test_kernel(CoreRef<MemSpace::device> const track_data,
                            DeviceCRef<UrbanMscData> const msc_data) {
  auto tid = KernelParamCalculator::thread_id();
  const celeritas::CoreTrackView track(track_data.params, track_data.states,
                                       tid);
  UrbanMsc msc{msc_data};

  auto sim = track.make_sim_view();
  auto step_limit = sim.step_limit();
  bool use_msc = msc.is_applicable(track, step_limit.step);
  if (use_msc) {
    msc.limit_step(track, &step_limit);
  }

  // Surrogate for propagation
  if (step_limit.step < 0.1) {
    step_limit.step = 0.1;
    step_limit.action = track.boundary_action();
  }
  if (use_msc) {
    msc.apply_step(track, &step_limit);
  }

  sim.force_step_limit(step_limit);
}
