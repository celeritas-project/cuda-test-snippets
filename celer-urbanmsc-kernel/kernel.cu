#include "hip/hip_runtime.h"

#include "celeritas/Types.hh"
#include "celeritas/em/data/UrbanMscData.hh"
#include "celeritas/em/msc/UrbanMsc.hh"
#include "celeritas/global/CoreTrackData.hh"
#include "celeritas/global/CoreTrackView.hh"
#include "celeritas/track/SimTrackView.hh"
#include "corecel/Macros.hh"
#include "corecel/Types.hh"
#include "corecel/sys/KernelParamCalculator.device.hh"
#include "corecel/sys/ThreadId.hh"

using namespace celeritas;
using namespace celeritas::detail;

__global__ void test_kernel(CoreRef<MemSpace::device> const track_data,
                            DeviceCRef<UrbanMscData> const msc_data) {
  auto tid = KernelParamCalculator::thread_id();
  const celeritas::CoreTrackView track(track_data.params, track_data.states,
                                       tid);
  UrbanMsc msc{msc_data};

  auto sim = track.make_sim_view();
  AlongStepLocalState local;
  local.step_limit = sim.step_limit();
  local.geo_step = local.step_limit.step;
  bool use_msc = msc.is_applicable(track, local.geo_step);
  if (use_msc) {
    msc.calc_step(track, &local);
  }

  // Surrogate for propagation
  if (local.geo_step < 0.1) {
    local.geo_step = 0.1;
    local.step_limit.action = track.boundary_action();
  }
  if (use_msc) {
    msc.apply_step(track, &local);
  } else {
    // Step might have been reduced due to geometry boundary
    local.step_limit.step = local.geo_step;
  }

  sim.force_step_limit(local.step_limit);
}
