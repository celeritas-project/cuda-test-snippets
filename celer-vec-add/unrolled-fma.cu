#include "hip/hip_runtime.h"
#include "celeritas/Types.hh"
#include "celeritas/phys/FourVector.hh"
#include "corecel/Macros.hh"
#include "corecel/Types.hh"
#include "corecel/math/ArrayOperators.hh"
#include "corecel/math/ArrayUtils.hh"

namespace celeritas {
__global__ void test_kernel(Real3 const* vscm_arr, Real3 const* vxcm_arr,
                              Real3 const* cm_dir_arr, FourVector const* fv_arr,
                              Real3* result_mom_arr) {
  Real3 const& vscm = vscm_arr[threadIdx.x];
  Real3 const& vxcm = vxcm_arr[threadIdx.x];
  Real3 const& cm_dir = cm_dir_arr[threadIdx.x];

  FourVector const& fv = fv_arr[threadIdx.x];
  Real3 result_mom;
  for (int i = 0; i < 3; ++i) {
    result_mom[i] = fv.mom[0] * vscm[i];
    result_mom[i] = std::fma(fv.mom[1], vxcm[i], result_mom[i]);
    result_mom[i] = std::fma(fv.mom[2], cm_dir[i], result_mom[i]);
  }
  result_mom_arr[threadIdx.x] = result_mom;
}
}  // namespace celeritas