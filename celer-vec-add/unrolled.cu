#include "hip/hip_runtime.h"
#include "celeritas/Types.hh"
#include "celeritas/phys/FourVector.hh"
#include "corecel/Macros.hh"
#include "corecel/Types.hh"
#include "corecel/math/ArrayOperators.hh"
#include "corecel/math/ArrayUtils.hh"

namespace celeritas {

__global__ void test_kernel_2(Real3 const* vscm_arr, Real3 const* vxcm_arr,
                              Real3 const* cm_dir_arr, FourVector const* fv_arr,
                              Real3* result_mom) {
  Real3 const& vscm = vscm_arr[threadIdx.x];
  Real3 const& vxcm = vxcm_arr[threadIdx.x];
  Real3 const& cm_dir = cm_dir_arr[threadIdx.x];

  FourVector fv = fv_arr[threadIdx.x];
  for (int i = 0; i < 3; ++i) {
    result_mom[threadIdx.x][i] =
        fv.mom[0] * vscm[i] + fv.mom[1] * vxcm[i] + fv.mom[2] * cm_dir[i];
  }
}

}  // namespace celeritas