
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void my_copysign(double* v) {
  int i = threadIdx.x;
  if (i & 1) {
    *v *= -1;
  }
}
