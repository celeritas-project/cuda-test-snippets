
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void my_copysign(double* v)
{
    int i = threadIdx.x; // assume threadIdx < 2
    *v = ((i << 1) - 1) * (*v);
}
