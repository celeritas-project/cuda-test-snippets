
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void my_copysign(double* v)
{
    int i = threadIdx.x;
    *v = (i == 0 ? 1 : -1) * (*v);
}
