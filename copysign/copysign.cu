
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void my_copysign(double* v)
{
    int i = threadIdx.x;
    *v = std::copysign(*v, double(i == 0 ? 1 : -1));
}
