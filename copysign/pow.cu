
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void my_copysign(double* v)
{
    int i = threadIdx.x;
    *v = std::pow(-1, i) * (*v);
}
