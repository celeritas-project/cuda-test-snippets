#include "hip/hip_runtime.h"
#include <cmath>

using real_type = double;
constexpr real_type fwhm_min_energy = 1e-3;

__global__ void loss_scaling(
    __restrict__ real_type const* energy,
    __restrict__ real_type* dst)
{
    real_type max_energy = energy[threadIdx.x];
    real_type result = real_type(0.5) * std::fmin(fwhm_min_energy / max_energy,
                                                  real_type(1))
           + real_type(1);
    dst[threadIdx.x] = result;
}
