#include "hip/hip_runtime.h"
#include <cmath>

using real_type = double;
constexpr real_type fwhm_min_energy = 1e-3;

__global__ void loss_scaling(
    __restrict__ real_type const* energy,
    __restrict__ real_type* dst)
{
    real_type max_energy = energy[threadIdx.x];
    real_type result = max_energy > fwhm_min_energy
              ? 1 + real_type(0.5) * fwhm_min_energy / max_energy
              : real_type(1.5);
    dst[threadIdx.x] = result;
}
