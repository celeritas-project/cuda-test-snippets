#include "hip/hip_runtime.h"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/SecondaryAllocatorPointers.hh"
#include "physics/base/SecondaryAllocatorView.hh"
#include "physics/em/EPlusGGInteractor.hh"
#include "random/cuda/RngEngine.cuh"

using namespace celeritas;

struct Params {
  ParticleParamsPointers particle;
  EPlusGGInteractorPointers interactor;
};

struct State {
  ParticleStatePointers particle;
  RngStatePointers rng;
  span<const Real3> direction;
  span<Interaction> result;
};

__global__ void test_kernel(Params const params, State const states,
                            SecondaryAllocatorPointers const secondaries) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  SecondaryAllocatorView allocate_secondaries(secondaries);
  ParticleTrackView particle(params.particle, states.particle, ThreadId(tid));
  RngEngine rng(states.rng, ThreadId(tid));

  EPlusGGInteractor interact(params.interactor, particle, states.direction[tid],
                             allocate_secondaries);

  Interaction result = interact(rng);

  states.result[tid] = result;
}
