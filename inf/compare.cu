
#include <hip/hip_runtime.h>
#include <cfloat>
#include <climits>
#include <cmath>

__global__ void isinf_kernel(const double* value, bool* result)
{
    result[threadIdx.x] = value[threadIdx.x] >= DBL_MAX;
}
