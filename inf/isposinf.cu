
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void isinf_kernel(const double* value, bool* result)
{
    result[threadIdx.x] = std::isinf(value[threadIdx.x]) & (value[threadIdx.x] > 0);
}
