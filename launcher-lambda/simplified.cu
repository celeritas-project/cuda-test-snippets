
#include <hip/hip_runtime.h>
struct MscData {
  float a;
  float b;
};

__global__ void apply_kernel(const MscData data, float const* __restrict__ step,
                             float* __restrict__ result) {
  result[threadIdx.x] = data.a * step[threadIdx.x] + data.b;
}