
#include <hip/hip_runtime.h>
struct MscData {
  float a;
  float b;
};

struct UrbanMsc {
  const MscData& data;

  __device__ auto make_calc_thing() const -> decltype(auto) {
    return [this](float step) { return this->data.a * step + this->data.b; };
  }
};

template <class F>
__device__ void apply_track(F calc_thing, float step, float* result) {
  *result = calc_thing(step);
}

__global__ void apply_kernel(const MscData data, float const* __restrict__ step,
                             float* __restrict__ result) {
  UrbanMsc msc{data};
  apply_track(msc.make_calc_thing(), step[threadIdx.x], &result[threadIdx.x]);
}