
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void myexp(float* value)
{
    value[threadIdx.x] = std::exp(value[threadIdx.x]);
}
