
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void myexp(double* value)
{
    value[threadIdx.x] = std::exp(value[threadIdx.x]);
}
