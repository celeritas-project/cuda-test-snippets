
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void mylog(double* value)
{
    value[threadIdx.x] = std::log(value[threadIdx.x]);
}
