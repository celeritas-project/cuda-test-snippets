
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void mylog(float* value)
{
    value[threadIdx.x] = std::log(value[threadIdx.x]);
}
