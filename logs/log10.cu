
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void mylog10(double* value)
{
    value[threadIdx.x] = std::log10(value[threadIdx.x]);
}
