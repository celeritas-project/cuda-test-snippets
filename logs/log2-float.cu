
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void mylog2(float* value)
{
    value[threadIdx.x] = std::log2(value[threadIdx.x]);
}
