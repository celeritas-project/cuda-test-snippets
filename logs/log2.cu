
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void mylog2(double* value)
{
    value[threadIdx.x] = std::log2(value[threadIdx.x]);
}
