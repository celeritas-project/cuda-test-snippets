
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void call_min(double* first, const double* second)
{
    first[threadIdx.x] = std::fmin(first[threadIdx.x], second[threadIdx.x]);
}
