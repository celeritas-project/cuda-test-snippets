
#include <hip/hip_runtime.h>
template<class T>
__device__ const T& mymin(const T& a, const T& b)
{
    return (b < a) ? b : a;
}

__global__ void call_min(double* first, const double* second)
{
    first[threadIdx.x] = mymin(first[threadIdx.x], second[threadIdx.x]);
}
