#include "hip/hip_runtime.h"
#include "shared.cuh"

__global__ void memset_zero(int* all_thread_ids)
{
    int i = thread_id();
    all_thread_ids[i] = 0;
}
