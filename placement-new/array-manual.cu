
#include <hip/hip_runtime.h>
#include <new>

struct Foo
{
    int value = 0x1234;
};

__global__ void kernel_independent(Foo* storage, Foo** initialized)
{
    Foo* start = storage + threadIdx.x * 2;
    initialized[threadIdx.x] = new (start) Foo;
    new (start + 1) Foo;
}
