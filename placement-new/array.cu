
#include <hip/hip_runtime.h>
#include <new>

struct Foo
{
    int value = 0x1234;
};

__global__ void kernel(Foo* storage, Foo** initialized)
{
    Foo* start = storage + threadIdx.x * 2;
    initialized[threadIdx.x] = new (start) Foo[2];
}
