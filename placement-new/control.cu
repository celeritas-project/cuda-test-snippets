
#include <hip/hip_runtime.h>
#include <new>

struct Foo
{
    int value = 0x1234;
};

__global__ void kernel_simple(Foo* storage, Foo** initialized)
{
    Foo* start = storage + threadIdx.x * 2;
    start[0].value = 0x1234;
    start[1].value = 0x1234;
    initialized[threadIdx.x] = start;
}
