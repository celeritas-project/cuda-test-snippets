
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void cbrt_pow(double* value)
{
    value[threadIdx.x] = std::pow(value[threadIdx.x], 1. / 3);
}
