
#include <hip/hip_runtime.h>
#include <cmath>
__global__ void cu_cbrt(double* value)
{
    value[threadIdx.x] = std::cbrt(value[threadIdx.x]);
}
