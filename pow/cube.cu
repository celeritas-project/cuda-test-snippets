
#include <hip/hip_runtime.h>
#include <cmath>

inline __device__ double cube(double value)
{
    return value * value * value;
}

__global__ void cu_cbrt(double* value)
{
    value[threadIdx.x] = cube(value[threadIdx.x]);
}
