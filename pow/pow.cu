
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void cu_pow(double* value)
{
    value[threadIdx.x] = std::pow(value[threadIdx.x], 3);
}
