
#include <hip/hip_runtime.h>
__global__ void print_values(const int* ints, const double* dbls,
                             int* result)
{
    int i = threadIdx.x;
    result[i] = ints[i] + (dbls[i] > 0.0);
}
