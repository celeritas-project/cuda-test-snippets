
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void print_values(const int* ints, const double* dbls,
                             int* result)
{
    int i = threadIdx.x;
    std::printf("Int %d, double %f\n", ints[i], dbls[i]);
    result[i] = ints[i] + (dbls[i] > 0.0);
}
