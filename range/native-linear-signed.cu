#include "hip/hip_runtime.h"
#include "base/Range.hh"

__global__ void move(const int max_steps, double distance, double* x) {
  auto start = (blockIdx.x * blockDim.x + threadIdx.x) * max_steps;
  for (int i = 0; i < max_steps; ++i) {
    x[start + i] += distance;
  }
}
