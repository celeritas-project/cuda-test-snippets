#include "hip/hip_runtime.h"
#include "base/Range.hh"

__global__ void move(const unsigned int max_steps, double distance, double* x) {
  auto start = (blockIdx.x * blockDim.x + threadIdx.x) * max_steps;
  for (unsigned int i = 0; i != max_steps; ++i) {
    x[start + i] += distance;
  }
}
