
#include <hip/hip_runtime.h>
__global__ void move(const int num_tracks, double distance, double* x) {
  for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < num_tracks;
       tid += blockDim.x * gridDim.x) {
    x[tid] += distance;
  }
}
