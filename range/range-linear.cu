#include "hip/hip_runtime.h"
#include "base/Range.hh"

__global__ void move(const unsigned int max_steps, double distance, double* x) {
  auto start = (blockIdx.x * blockDim.x + threadIdx.x) * max_steps;
  for (auto i : celeritas::range(max_steps)) {
    x[start + i] += distance;
  }
}
