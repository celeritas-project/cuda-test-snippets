#include "hip/hip_runtime.h"
#include "base/Range.hh"

__global__ void move(const unsigned int num_tracks, double distance, double* x) {
  for (auto tid :
       celeritas::range(blockIdx.x * blockDim.x + threadIdx.x, num_tracks)
           .step(blockDim.x * gridDim.x)) {
    x[tid] += distance;
  }
}
