
#include <hip/hip_runtime.h>
__global__ void select(const double* ran, const double* total, const int* size,
                       double** frac, double** xs, int* selection) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  double cutoff = total[idx] * ran[idx];
  const double* el_frac = frac[idx];
  const double* el_xs = xs[idx];
  unsigned int i = size[idx] - 1;
  for (; i > 0; --i) {
    cutoff -= el_frac[i] * el_xs[i];
    if (cutoff <= 0)
        break;
  }
  selection[idx] = i;
}
