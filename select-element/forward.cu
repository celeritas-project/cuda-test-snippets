
#include <hip/hip_runtime.h>
__global__ void select(const double* ran, const double* total, const int* size,
                       double** frac, double** xs, int* selection) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  double cutoff = total[idx] * ran[idx];
  double accum = 0;
  const double* el_frac = frac[idx];
  const double* el_xs = xs[idx];
  int el_size = size[idx];
  for (int i = 0; i < el_size - 1; ++i) {
    accum += el_frac[i] * el_xs[i];
    if (accum >= cutoff) selection[idx] = i;
  }
  selection[idx] = el_size - 1;
}
