
#include <hip/hip_runtime.h>
__global__ void select(const double* ran, const double* total, const int* size,
                       double** frac, double** xs, int* selection) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  double cutoff = total[idx] * ran[idx];
  double accum = 0;
  const double* el_frac = frac[idx];
  const double* el_xs = xs[idx];
  unsigned int i = 0;
  unsigned int imax = size[idx] - 1;
  for (; i != imax; ++i) {
    accum += el_frac[i] * el_xs[i];
    if (cutoff >= accum)
        break;
  }
  selection[idx] = i;
}
