
#include <hip/hip_runtime.h>
__global__ void select(const double* ran, const double* total, const int* size,
                       double** frac, double** xs, int* selection) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  double accum = -total[idx] * ran[idx];
  const double* el_frac = frac[idx];
  const double* el_xs = xs[idx];
  unsigned int i = 0;
  unsigned int imax = size[idx] - 1;
  for (; i != imax; ++i) {
    accum += el_frac[i] * el_xs[i];
    if (accum >= 0)
        break;
  }
  selection[idx] = i;
}
