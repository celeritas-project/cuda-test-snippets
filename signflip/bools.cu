
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void conditional(double* __restrict__ out,
                            double const* __restrict__ in,
                            double const* __restrict__ sgn) {
  int i = threadIdx.x;
  bool is_positive = sgn[i] > 0;
  out[i] = in[i] * (2 * is_positive - 1);
}
