
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void conditional(double* __restrict__ out,
                           double const* __restrict__ in,
                           double const* __restrict__ sgn) {
  int i = threadIdx.x;
  double helicity = sgn[i] > 0 ? 1 : -1;
  out[i] = in[i] * helicity;
}
