
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void conditional(double* __restrict__ out,
                           double const* __restrict__ in,
                           double const* __restrict__ sgn) {
  int i = threadIdx.x;
  bool is_pos = sgn[i] > 0;
  out[i] = is_pos ? in[i] : -in[i];
}
