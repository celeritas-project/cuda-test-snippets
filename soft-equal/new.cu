
#include <hip/hip_runtime.h>
#include <cmath>
__device__ __forceinline__ bool soft_equal(double a, double b, double abs, double rel) {
  double thresh = std::fmax(abs, rel * std::fmax(std::fabs(a), std::fabs(b)));
  return std::fabs(a - b) < thresh;
}

__global__ void soft_equal_kernel(bool* dest, double* a, double* b,
                                  double abs, double rel) {
  dest[threadIdx.x] = soft_equal(a[threadIdx.x], b[threadIdx.x], abs, rel);
}
