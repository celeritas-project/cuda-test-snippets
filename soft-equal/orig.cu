
#include <hip/hip_runtime.h>
#include <cmath>
__device__ __forceinline__ bool soft_equal_orig(double a, double b, double abs, double rel) {
  const double abs_e = std::fabs(a);
  if (std::fabs(b - a) < rel * abs_e) {
    return true;
  }
  const double eps_abs = abs;
  const double abs_a = std::fabs(b);
  if ((abs_e < eps_abs) && (abs_a < rel)) {
    return true;
  }
  if ((abs_a < eps_abs) && (abs_e < rel)) {
    return true;
  }
  if (std::isinf(a) && std::isinf(b) && std::signbit(a) == std::signbit(b)) {
    return true;
  }

  return false;
}

__global__ void soft_equal_kernel(bool* dest, double* a, double* b,
                                  double abs, double rel) {
  dest[threadIdx.x] = soft_equal_orig(a[threadIdx.x], b[threadIdx.x], abs, rel);
}
