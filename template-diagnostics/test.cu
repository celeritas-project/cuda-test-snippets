
#include <hip/hip_runtime.h>
#include <iostream>
using std::cout; using std::endl;

template <class T>
__global__ void fill(T* dst) {
  dst[threadIdx.x] = 0;
}

template<class F>
int get_num_reg(F* func)
{
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<void const*>(func)));
    return attr.numRegs;
}

template<class F>
struct Executor
{
    F launch_kernel;

    template<class... Ts>
    void operator()(Ts... args)
    {
        cout << "num reg: " << get_num_reg(launch_kernel) << endl;
        launch_kernel<<<256, 1, 1>>>(args...);
    }
};


template<class F>
decltype(auto) make_executor(F* func)
{
    return Executor<F*>{func};
}

int main()
{
    int* device_data{nullptr};
    hipMalloc(&device_data, 256);

    auto launch = make_executor(fill<int>);
    launch(device_data);
    return 0;
}