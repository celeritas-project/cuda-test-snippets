
#include <hip/hip_runtime.h>
struct Point {
  double* x;
  double* y;
  double* z;
};

struct Ref {
  Point pos;
  Point dir;
  double* distance;
};

struct View {
  int size;
  double3* pos;
  double3* dir;
  double* distance;

  __device__ Ref operator[](int i) const {
    double3* newpos = pos + i;
    double3* newdir = dir + i;
    return {{&(newpos->x), &(newpos->y), &(newpos->z)},
            {&(newdir->x), &(newdir->y), &(newdir->z)},
            distance + i};
  }
};

__device__ inline void move_impl(const Ref& ref) {
  const double nextdist = *ref.distance;
  *ref.pos.x += *ref.dir.x * nextdist;
  *ref.pos.y += *ref.dir.y * nextdist;
  *ref.pos.z += *ref.dir.z * nextdist;
}

__global__ void move(View view) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < view.size) {
    move_impl(view[idx]);
  }
}
