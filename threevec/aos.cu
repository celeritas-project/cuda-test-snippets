
#include <hip/hip_runtime.h>
using Point = double3;

struct Ref {
  Point* pos;
  Point* dir;
  double* distance;
};

struct View {
  int size;
  Point* pos;
  Point* dir;
  double* distance;

  __device__ Ref operator[](int i) const {
    return {pos + i, dir + i, distance + i};
  }
};

__device__ inline void move_impl(const Ref& ref) {
  const double nextdist = *ref.distance;
  ref.pos->x += ref.dir->x * nextdist;
  ref.pos->y += ref.dir->y * nextdist;
  ref.pos->z += ref.dir->z * nextdist;
}

__global__ void move(View view) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < view.size) {
    move_impl(view[idx]);
  }
}
