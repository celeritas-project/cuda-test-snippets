
#include <hip/hip_runtime.h>
struct Point {
  double* x;
  double* y;
  double* z;
};

struct Ref {
  Point pos;
  Point dir;
  double* distance;
};

struct View {
  int size;
  double* pos;
  double* dir;
  double* distance;

  __device__ Ref operator[](int i) const {
    return {{pos + i, pos + i + size, pos + i + 2*size},
            {dir + i, dir + i + size, dir + i + 2*size},
            distance + i};
  }
};

__device__ inline void move_impl(const Ref& ref) {
  const double nextdist = *ref.distance;
  *ref.pos.x += *ref.dir.x * nextdist;
  *ref.pos.y += *ref.dir.y * nextdist;
  *ref.pos.z += *ref.dir.z * nextdist;
}

__global__ void move(View view) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < view.size) {
    move_impl(view[idx]);
  }
}
