
#include <hip/hip_runtime.h>
struct Point {
  double* x;
  double* y;
  double* z;
};

struct Ref {
  Point pos;
  Point dir;
  double* distance;
};

struct View {
  int size;
  Point pos;
  Point dir;
  double* distance;

  __device__ Ref operator[](int i) const {
    return {{pos.x + i, pos.y + i, pos.z + i},
            {dir.x + i, dir.y + i, dir.z + i},
            distance + i};
  }
};

__device__ inline void move_impl(const Ref& ref) {
  const double nextdist = *ref.distance;
  *ref.pos.x += *ref.dir.x * nextdist;
  *ref.pos.y += *ref.dir.y * nextdist;
  *ref.pos.z += *ref.dir.z * nextdist;
}

__global__ void move(View view) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < view.size) {
    move_impl(view[idx]);
  }
}
