
#include <hip/hip_runtime.h>
struct Real3
{
    double value[3];
};

__device__ void copy(const Real3& in1, const Real3& in2,
                          Real3* out1, Real3* out2)
{
    *out1 = in1;
    *out2 = in2;
}

__global__ void call_min(int* offsets, const Real3* inputs, Real3* outputs)
{
    int idx = offsets[threadIdx.x];
    // Copy with some bogus offsets
    copy(inputs[idx], inputs[idx + 1], &outputs[idx - 1], &outputs[idx]);
}
