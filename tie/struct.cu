
#include <hip/hip_runtime.h>
struct Real3
{
    double value[3];
};

struct ReturnType
{
    Real3 first, second;
};

__device__ ReturnType copy(const Real3& in1, const Real3& in2)
{
    return {in1, in2};
}

__global__ void call_min(int* offsets, const Real3* inputs, Real3* outputs)
{
    int idx = offsets[threadIdx.x];
    // Copy with some bogus offsets
    auto result = copy(inputs[idx], inputs[idx + 1]);
    outputs[idx - 1] = result.first;
    outputs[idx] = result.second;
}
