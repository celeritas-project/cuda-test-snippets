
#include <hip/hip_runtime.h>
struct Real3
{
    double value[3];
};

template<class T1, class T2>
struct Pair
{
    T1 first;
    T2 second;

    template<class U1, class U2>
    __device__ Pair& operator=(const Pair<U1, U2>& other)
    {
        first = other.first;
        second = other.second;
        return *this;
    }
};

template<class T1, class T2>
inline __device__ Pair<T1&,T2&> pair_tie(T1& first, T2& second)
{
    return {first, second};
}

__device__ Pair<Real3, Real3> copy(const Real3& in1, const Real3& in2)
{
    return {in1, in2};
}

__global__ void call_min(int* offsets, const Real3* inputs, Real3* outputs)
{
    int idx = offsets[threadIdx.x];
    // Copy with some bogus offsets
    pair_tie(outputs[idx - 1], outputs[idx])
        = copy(inputs[idx], inputs[idx + 1]);
}
