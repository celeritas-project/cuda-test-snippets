#include "hip/hip_runtime.h"
#include "shared.cuh"

struct ParticleRef {
  Point pos;
  Point dir;
  double nextdist;
};

inline __device__ ParticleRef make_ref(const ParticleView &view, int i) {
  return {view.get_pos(i), view.get_dir(i), view.get_nextdist(i)};
}

__device__ inline void move_impl(const ParticleRef ref) {
  const double x = *ref.pos.x;
  const double y = *ref.pos.y;
  const double z = *ref.pos.z;
  const double u = *ref.dir.x;
  const double v = *ref.dir.y;
  const double w = *ref.dir.z;
  *ref.pos.x = x + u * ref.nextdist;
  *ref.pos.y = y + v * ref.nextdist;
  *ref.pos.z = z + w * ref.nextdist;
}

__global__ void move(ParticleView view) {
  int i = thread_id();
  if (i >= view.size) return;
  move_impl(make_ref(view, i));
}
