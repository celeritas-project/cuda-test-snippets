#include "hip/hip_runtime.h"
#include "shared.cuh"

struct ParticleRef {
  Point pos;
  Point dir;
  double nextdist;
};

inline __device__ ParticleRef make_ref(const ParticleView &view, int i) {
  return {view.get_pos(i), view.get_dir(i), view.get_nextdist(i)};
}

__device__ inline void saxpy(double *__restrict__ x, double *__restrict__ y,
                             double *__restrict__ z,
                             const double *__restrict__ u,
                             const double *__restrict__ v,
                             const double *__restrict__ w, double distance) {
  *x += *u * distance;
  *y += *v * distance;
  *z += *w * distance;
}

__device__ inline void move_impl(const ParticleRef ref) {
  saxpy(ref.pos.x, ref.pos.y, ref.pos.z, ref.dir.x, ref.dir.y, ref.dir.z,
        ref.nextdist);
}

__global__ void move(ParticleView view) {
  int i = thread_id();
  if (i >= view.size) return;
  move_impl(make_ref(view, i));
}
