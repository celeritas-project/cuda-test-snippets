#include "hip/hip_runtime.h"
#include "shared.cuh"

struct ParticleRef {
  Point pos;
  Point dir;
  double nextdist;
};

inline __device__ ParticleRef make_ref(const ParticleView &view, int i) {
  return {view.get_pos(i), view.get_dir(i), view.get_nextdist(i)};
}

__device__ inline void move_impl(const ParticleRef ref) {
  *ref.pos.x += *ref.dir.x * ref.nextdist;
  *ref.pos.y += *ref.dir.y * ref.nextdist;
  *ref.pos.z += *ref.dir.z * ref.nextdist;
}

__global__ void move(ParticleView view) {
  int i = thread_id();
  if (i >= view.size) return;
  move_impl(make_ref(view, i));
}
