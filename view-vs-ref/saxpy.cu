#include "hip/hip_runtime.h"
#include "shared.cuh"

__global__ void move(int size, double *__restrict__ x, double *__restrict__ y,
                     double *__restrict__ z, const double *__restrict__ u,
                     const double *__restrict__ v, const double *__restrict__ w,
                     const double *__restrict__ distance) {
  int i = thread_id();
  if (i >= size) return;
  x[i] += u[i] * distance[i];
  y[i] += v[i] * distance[i];
  z[i] += w[i] * distance[i];
}
