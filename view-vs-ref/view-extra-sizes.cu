#include "hip/hip_runtime.h"
#include "shared.cuh"

template <class T>
struct View {
  int size;
  T *ptr;
};

struct SizedParticleView {
  View<double> x, y, z, u, v, w, nextdist;

  __device__ int size() const { return x.size; }

  __device__ Point get_pos(int i) const {
    return {x.ptr + i, y.ptr + i, z.ptr + i};
  }
  __device__ Point get_dir(int i) const {
    return {u.ptr + i, v.ptr + i, w.ptr + i};
  }
  __device__ double get_nextdist(int i) const { return nextdist.ptr[i]; }
};

__device__ inline void move_impl(const SizedParticleView &view, int i) {
  *view.get_pos(i).x += *view.get_dir(i).x * view.get_nextdist(i);
  *view.get_pos(i).y += *view.get_dir(i).y * view.get_nextdist(i);
  *view.get_pos(i).z += *view.get_dir(i).z * view.get_nextdist(i);
}

__global__ void move(SizedParticleView view) {
  int i = thread_id();
  if (i >= view.size()) return;
  move_impl(view, i);
}
