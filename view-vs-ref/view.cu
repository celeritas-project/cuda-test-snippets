#include "hip/hip_runtime.h"
#include "shared.cuh"

__device__ inline void move_impl(const ParticleView &view, int i) {
  *view.get_pos(i).x += *view.get_dir(i).x * view.get_nextdist(i);
  *view.get_pos(i).y += *view.get_dir(i).y * view.get_nextdist(i);
  *view.get_pos(i).z += *view.get_dir(i).z * view.get_nextdist(i);
}

__global__ void move(ParticleView view) {
  int i = thread_id();
  if (i >= view.size) return;
  move_impl(view, i);
}
